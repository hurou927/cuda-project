#include "hip/hip_runtime.h"
//
// cooperative_group
//   -> nvcc ... -rdc = true  and require TCC mode
//
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#include "header/my_gettime.hpp"
#include "header/my_cuda_device.cuh"
#include "header/my_cuda_host.cuh"
#include "header/fp16_dev.h"

#define NUMTHREADS 128

__global__ void kernel(const int* __restrict__ GIN, int *GOUT){
	int laneId = threadIdx.x & 0x1F;
	int warpId = threadIdx.x >> 5;
	int tx = blockDim.x*blockIdx.x+threadIdx.x;

	__shared__ int s_mem[NUMTHREADS];
	s_mem[threadIdx.x]=GIN[tx];
	__syncthreads();
	int val = s_mem[threadIdx.x];
	val = warp_scan<32>(val,laneId);
	val = warp_sum(val,laneId);
	val = block_scan<int, NUMTHREADS>(val,warpId,laneId);
	val = block_sum <int, NUMTHREADS>(val,warpId,laneId);
	GOUT[tx]=val;



	__threadfence();
}

void execGPUkerenel(){
	hipProfilerStart();
	cudatimeStamp ts(10);
	printf("%d\n",CUDART_VERSION);
	int numthreads= NUMTHREADS;
	int numblocks = 256;

	size_t num_items = (size_t) numblocks*numthreads;
	//host memory
	int *h_in=(int *)malloc(sizeof(int)*num_items);
	int *h_out=(int *)malloc(sizeof(int)*num_items);
	for(int i=0;i<num_items;i++) h_in[i]=i;
	//device memory
	int *d_in;
	hipMalloc((void **)&d_in,sizeof(int)*num_items);
	int *d_out;
	hipMalloc((void **)&d_out,sizeof(int)*num_items);

	// Change dynamic Shared memory size (CC7.0~)
	//int maxbytes = 98304; // 96 KB 
	//hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

	//error check
	checkCudaStatus();
	//memcpy Host->Device
	ts.stamp();
	hipMemcpy(d_in,h_in,sizeof(int)*num_items,hipMemcpyHostToDevice);
	// memset
	ts.stamp();
	hipMemset(d_out,0,sizeof(int)*num_items);
	//kernel
	ts.stamp();
	// cooperativeKernel (Pascal~)
	// void **args = { (void*)&d_in , (void*)&d_out  };
	// hipLaunchCooperativeKernel(reinterpret_cast<const void*>(kernel), numblocks, numthreads, args);
	kernel <<< numblocks , numthreads >>> (d_in,d_out);

	ts.stamp();
	//memcpy Device->Host
	hipMemcpy(h_out,d_out,sizeof(int)*num_items,hipMemcpyDeviceToHost);
	ts.stamp();

	ts.print();//ts.print_hori();
	printCudaLastError();

	printf("occupancy,%4.3f,SMcount,%d,activeblock,%d\n",occupancy(kernel,NUMTHREADS),get_sm_count(),get_activeblock_per_device(kernel,NUMTHREADS));

	//memory free
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);

	hipProfilerStop();

}

int main(int argc,char **argv){
	
	GPUBoost(1);

	execGPUkerenel();

	return 0;
}

#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#include "my_gettime.hpp"
#include "my_cuda_device.cuh"
#include "my_cuda_host.cuh"

#define NUMTHREADS 128

__global__ void kernel(const int* __restrict__ IN, int *OUT){
	int laneId = threadIdx.x & 0x1F;
	int warpId = threadIdx.x >> 5;
	int tx = blockDim.x*blockIdx.x+threadIdx.x;

	__shared__ int s_mem[NUMTHREADS];
	s_mem[threadIdx.x]=IN[tx];
	__syncthreads();
	int val = s_mem[threadIdx.x];
	val = warp_scan<32>(val,laneId);
	val = warp_sum(val,laneId);
	val = block_scan<int, NUMTHREADS>(val,warpId,laneId);
	val = block_sum <int, NUMTHREADS>(val,warpId,laneId);
	OUT[tx]=val;
}

int main(int argc,char **argv){
	hipProfilerStart();
	cudatimeStamp ts(10);

	int numthreads= NUMTHREADS;
	int numblocks = 256;
	
	size_t num_items = (size_t) numblocks*numthreads;
	//host memory
	int *h_in=(int *)malloc(sizeof(int)*num_items);
	int *h_out=(int *)malloc(sizeof(int)*num_items);
	for(int i=0;i<num_items;i++) h_in[i]=i;
	//device memory
	int *d_in;
	hipMalloc((void **)&d_in,sizeof(int)*num_items);
	int *d_out;
	hipMalloc((void **)&d_out,sizeof(int)*num_items);

	//error check
	checkCudaStatus();
	//memcpy Host->Device
	ts.stamp();
	hipMemcpy(d_in,h_in,sizeof(int)*num_items,hipMemcpyHostToDevice);
	// memset
	ts.stamp();
	hipMemset(d_in,0,sizeof(int)*num_items);
	//kernel
	ts.stamp();
	kernel <<< numblocks , numthreads >>> (d_in,d_out);

	ts.stamp();
	//memcpy Device->Host
	hipMemcpy(h_out,d_out,sizeof(int)*num_items,hipMemcpyDeviceToHost);
	ts.stamp();

	ts.print();//ts.print_hori();
	printCudaLastError();

	printf("occupancy,%4.3f,SMcount,%d,activeblock,%d\n",occupancy(kernel,NUMTHREADS),get_sm_count(),get_activeblock_per_device(kernel,NUMTHREADS));

	//memory free
	//free(h);
	//hipFree(d);

	hipProfilerStop();
	return 0;
}
